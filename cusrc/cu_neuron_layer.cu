/*
 * File description: cu_neuron_layer.cu
 * Author information: Mike Ranzinger mranzinger@alchemyapi.com
 * Copyright information: Copyright Orchestr8 LLC
 */

#include "cu_neuron_layer.cuh"

#include <stdexcept>

#include "cumat.cuh"
#include "cumath_unary_functions.cuh"
#include "cumath_binary_functions.cuh"

using namespace std;

template<typename UnaryFn>
void NCompute(const Params &input, Params &output, UnaryFn fn, hipblasHandle_t handle)
{
    input.GetCudaMatrix(handle).UnaryExpr<false>(output.GetCudaMatrix(handle), fn);
}

void NCompute(const Params &input, Params &output, CuIdentity, hipblasHandle_t handle)
{
    output = input;
}


void NBackprop(const Params &lastInput, const Params &lastOutput, Params &inputErrs,
               CuIdentity, hipblasHandle_t handle)
{
    inputErrs.GetCudaMatrix(handle).SetConstant(1.0f);
}

void NBackprop(const Params &lastInput, const Params &lastOutput, Params &inputErrs,
               CuLogistic, hipblasHandle_t handle)
{
    lastOutput.GetCudaMatrix(handle).UnaryExpr<false>(inputErrs.GetCudaMatrix(handle),
            CuLogisticDerivativeCalc());
}

void NBackprop(const Params &lastInput, const Params &lastOutput, Params &inputErrs,
               CuRectifier, hipblasHandle_t handle)
{
    lastInput.GetCudaMatrix(handle).UnaryExpr<false>(inputErrs.GetCudaMatrix(handle),
            CuRectifierDerivative());
}

void NBackprop(const Params &lastInput, const Params &lastOutput, Params &inputErrs,
               CuTanh, hipblasHandle_t handle)
{
    lastOutput.GetCudaMatrix(handle).UnaryExpr<false>(inputErrs.GetCudaMatrix(handle),
            CuTanhDerivativeCalc());
}

void NBackprop(const Params &lastInput, const Params &lastOutput, Params &inputErrs,
                CuRamp, hipblasHandle_t handle)
{
    lastInput.GetCudaMatrix(handle).UnaryExpr<false>(inputErrs.GetCudaMatrix(handle),
            CuRampDerivative());
}

void NBackprop(const Params &lastInput, const Params &lastOutput, Params &inputErrs,
            CuSoftplus, hipblasHandle_t handle)
{
    lastInput.GetCudaMatrix(handle).UnaryExpr<false>(inputErrs.GetCudaMatrix(handle),
            CuSoftplusDerivativeRaw());
}

void NBackprop(const Params &lastInput, const Params &lastOutput, Params &inputErrs,
            CuHardTanh, hipblasHandle_t handle)
{
    lastInput.GetCudaMatrix(handle).UnaryExpr<false>(inputErrs.GetCudaMatrix(handle),
            CuHardTanhDerivative());
}

template<typename CalcFn>
class CuNeuronLayer
    : public ICuNeuronLayer
{
private:
    int _deviceId;
    hipblasHandle_t _handle;

public:
    CuNeuronLayer(int deviceId)
        : _deviceId(deviceId), _handle(0)
    {
        // TODO: Get handle
    }

    virtual Params Compute(const Params &input, bool isTraining)
    {
        CuMat *m = new CuMat(_handle, input.Rows, input.Cols);

        Params ret(input, m);

        NCompute(input, ret, CalcFn(), _handle);

        return ret;
    }
    virtual Params Backprop(const Params &input, const Params &lastOutput,
                            const Params &outputErrors)
    {
        CuMat *inputErrs = new CuMat(_handle, input.Rows, input.Cols);

        Params ret(input, inputErrs);

        NBackprop(input, lastOutput, ret, CalcFn(), _handle);

        inputErrs->CoeffMultiply(outputErrors.GetCudaMatrix(_handle));

        return ret;
    }
};

ICuNeuronLayer* CreateCuNeuronLayer(int deviceId, CuNeuronType type)
{
    switch (type)
    {
    case Cut_Linear:
        return new CuNeuronLayer<CuIdentity>(deviceId);
    case Cut_Logistic:
        return new CuNeuronLayer<CuLogistic>(deviceId);
    case Cut_Rectifier:
        return new CuNeuronLayer<CuRectifier>(deviceId);
    case Cut_Tanh:
        return new CuNeuronLayer<CuTanh>(deviceId);
    case Cut_Ramp:
        return new CuNeuronLayer<CuRamp>(deviceId);
    case Cut_SoftPlus:
        return new CuNeuronLayer<CuSoftplus>(deviceId);
    case Cut_HardTanh:
        return new CuNeuronLayer<CuHardTanh>(deviceId);
    default:
        throw runtime_error("Invalid neuron type.");
    }
}





















