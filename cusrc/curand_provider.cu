#include "hip/hip_runtime.h"

#include "curand_provider.cuh"

#include <stdexcept>
#include <sys/time.h>

#include <hip/hip_runtime.h>

#include "cumath_functions.cuh"

using namespace std;

CURandProvider *CURandProvider::s_instance = NULL;

CURandProvider::CURandProvider()
{
}

CURandProvider::~CURandProvider()
{
}

hiprandState* CURandProvider::GetRandomStates(int device, uint32_t requiredLen)
{
    return GetInstance()->p_GetRandomStates(device, requiredLen);
}

hiprandState* CURandProvider::GetRandomStates(int device, dim3 gridDim, dim3 blockDim)
{
    return GetInstance()->p_GetRandomStates(device, gridDim, blockDim);
}

CURandProvider* CURandProvider::GetInstance()
{
    if (s_instance == NULL)
        s_instance = new CURandProvider();

    return s_instance;
}

__global__ void cudaInitRandoms(hiprandState *state, uint64_t seed, uint32_t maxId)
{
    uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= maxId)
        return;

    hiprand_init(seed, id, 0, state + id);
}

hiprandState* CURandProvider::p_GetRandomStates(int device, uint32_t requiredLen)
{
    CURandDeviceInfo &devInfo = m_buffers[device];

    if (devInfo.ArrayLen >= requiredLen)
        return devInfo.States;

    bool tryPreserve = devInfo.States != NULL;
    hiprandState *newBuff = NULL;

    hipError_t err = hipMalloc(&newBuff, requiredLen * sizeof(hiprandState));

    if (tryPreserve && err != hipSuccess)
    {
        // Free the old buffer I suppose. This means that a full re-init will need
        // to be performed
        hipFree(devInfo.States);
        devInfo.ArrayLen = 0;
        devInfo.States = NULL;

        tryPreserve = false;

        err = hipMalloc(&newBuff, requiredLen * sizeof(hiprandState));
    }

    if (err != hipSuccess)
        throw runtime_error("Unable to allocate random buffer large enough to satisfy request.");

    uint32_t offset = 0;

    if (tryPreserve)
    {
        hipMemcpy(newBuff, devInfo.States, devInfo.ArrayLen * sizeof(hiprandState),
                   hipMemcpyDeviceToDevice);
        hipFree(devInfo.States);

        offset = devInfo.ArrayLen;
    }

    devInfo.ArrayLen = requiredLen;
    devInfo.States = newBuff;

    // Initialize all of the new random states
    cudaInitRandoms
        <<<round_up(requiredLen - offset, 128), 128>>>
            (devInfo.States + offset,
             time(NULL),
             requiredLen - offset);

    return devInfo.States;
}

hiprandState* CURandProvider::p_GetRandomStates(int device, dim3 gridDim, dim3 blockDim)
{
    uint32_t reqLen = (gridDim.z * blockDim.z) *
                      (gridDim.y * blockDim.y) *
                      (gridDim.x * blockDim.x);

    return p_GetRandomStates(device, reqLen);
}


