/*
 * cusetup_provider.cu
 *
 *  Created on: Jun 4, 2014
 *      Author: mike
 */

#include "cusetup_provider.cuh"

#include <iostream>

using namespace std;

CuSetupProvider::CuSetupProvider()
{
}

CuSetupProvider::~CuSetupProvider()
{
	/*for (HandleMap::iterator iter = _handleMap.begin(),
			                 end = _handleMap.end();
			iter != end;
			++iter)
	{
	    hipError_t err = hipSetDevice(iter->second.Device);
	    if (err != hipSuccess)
	    {
	        cerr << "Failed to set the device before freeing. Error: " << err << endl;
	        exit(1);
	    }

	    err = hipDeviceSynchronize();
	    if (err != hipSuccess)
	    {
	        cerr << "Failed to synchronize with the device before freeing the cublas handle. Error: " << err << endl;
	        exit(1);
	    }

		hipblasStatus_t status = hipblasDestroy(iter->second.CublasHandle);
		if (status != HIPBLAS_STATUS_SUCCESS)
		{
		    cerr << "Failed to destroy the cublas handle. Error: " << status << endl;
		}
	}*/
}

CuContext CuSetupProvider::GetHandle(int deviceId, int threadId)
{
	return Instance().p_GetHandle(deviceId, threadId);
}

CuSetupProvider& CuSetupProvider::Instance()
{
	static CuSetupProvider s_provider;

	return s_provider;
}

CuContext CuSetupProvider::p_GetHandle(int deviceId, int threadId)
{
	std::pair<int, int> key(deviceId, threadId);

	HandleMap::iterator iter = _handleMap.find(key);

	if (iter != _handleMap.end())
		return iter->second;

	CuContext &ret = _handleMap[key];
	ret.Device = deviceId;

	hipSetDevice(deviceId);

	hipblasCreate(&ret.CublasHandle);

	return ret;
}
