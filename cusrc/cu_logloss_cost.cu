#include "hip/hip_runtime.h"
/*
 * cu_logloss_cost.cu
 *
 *  Created on: Jun 8, 2014
 *      Author: mike
 */

#include "cu_logloss_cost.cuh"

#include "cusetup_provider.cuh"
#include "cumat.cuh"

__device__ __constant__ Real s_epss = 0.000000001;

#define safe_pred(val) min(max(val, s_epss), 1.0f - s_epss)

CuLoglossCost::CuLoglossCost(int deviceId)
	: _outputIsSoftmax(false)
{
	_handle = CuSetupProvider::GetHandle(deviceId);

	_secondHandle.Device = deviceId;
	hipblasCreate(&_secondHandle.CublasHandle);

	hipStreamCreate(&_secondStream);

	_cacheCompLL = new CuMat(_handle);
	_cacheCompMaxIdxs = new CuMat(_secondHandle);
	_cacheCompBinarized = new CuMat(_secondHandle);

	_cacheCost = new CuMat(_handle);
	_cacheCost->SetSharedModify(true);
}

CuLoglossCost::~CuLoglossCost()
{
	delete _cacheCompLL;
	delete _cacheCompMaxIdxs;
	delete _cacheCompBinarized;
	delete _cacheCost;

	hipStreamDestroy(_secondStream);

	hipblasDestroy(_secondHandle.CublasHandle);
}

struct CuLLVecComputeFn
{
	__device__ Real operator()(Real pred, Real label) const
	{
		Real sPred = safe_pred(pred);

		return label * log(sPred) + (1.0f - label) * log(1 - sPred);
	}
};

struct CuLLVecGradFn
{
	const Real _scale;

	CuLLVecGradFn(const CuMat &labMat)
		: _scale(1.0f / labMat.Cols()) { }

	__device__ Real operator()(Real pred, Real label) const
	{
		Real sPred = safe_pred(pred);

		return (((1.0f - label) / (1.0f - sPred)) - (label / sPred)) * _scale;
	}
};

struct CuLLIdxComputeFn
{
	const Real *_labMat;

	CuLLIdxComputeFn(const CuMat &labMat)
		: _labMat(labMat.Buff()) { }

	__device__ Real operator()(Real pred, uint32_t row, uint32_t col) const
	{
		Real sPred = safe_pred(pred);

		Real labIdx = _labMat[col];

		if (col == labIdx)
			return log(sPred);
		else
			return log(1.0f - sPred);
	}
};

struct CuLLIdxGradFn
{
	const Real _scale;
	const Real *_labMat;

	CuLLIdxGradFn(const CuMat &labMat)
		: _labMat(labMat.Buff()), _scale(1.0f / labMat.Cols()) { }

	__device__ Real operator()(Real pred, uint32_t row, uint32_t col) const
	{
		Real sPred = safe_pred(pred);

		Real labIdx = _labMat[col];

		if (col == labIdx)
		{
			return (-1.0f / sPred) * _scale;
		}
		else
		{
			return (1.0f / (1.0f - sPred)) * _scale;
		}
	}
};

struct CuLLVecMaxEqFn
{
	const Real *_maxIdx;

	CuLLVecMaxEqFn(const CuMat &maxIdxMat)
		: _maxIdx(maxIdxMat.Buff()) { }

	__device__ Real operator()(Real pred, Real label, uint32_t row, uint32_t col) const
	{
		if (label == 0.0f)
			return 0.0f;

		Real mIdx = _maxIdx[col];

		if (mIdx == row)
			return 1.0f;
		else
			return 0.0f;
	}
};

struct CuLLIdxMaxEqFn
{
	__device__ Real operator()(Real maxIdx, Real labelIdx) const
	{
		return maxIdx == labelIdx;
	}
};

struct CuLLIdxSoftmaxGradFn
{
	const Real *_pLabels;
	const Real _scale;

	CuLLIdxSoftmaxGradFn(const CuMat &labels)
		: _pLabels(labels.Buff()),
		  _scale(1.0f / labels.Cols()) { }

	__device__ Real operator()(Real pred, uint32_t row, uint32_t col) const
	{
		const Real label = _pLabels[col];

		const Real val = (row == label) ? pred - 1.0f : pred;

		const Real scaled = _scale * val;

		return scaled;
	}
};

CostMap CuLoglossCost::Compute(const Params& pred, const Params& labels)
{
	const CuMat &mPred = pred.GetCudaMatrix(_handle);
	const CuMat &mLabels = labels.GetCudaMatrix(_handle);

	Real logLoss, numCorr;

	hipStreamSynchronize(0);

	// Get the index for the maximum value in each column
	mPred.Colwise().MaxIdx(*_cacheCompMaxIdxs);

	// Index mode. Each label is stored by index
	if (labels.Rows == 1)
	{
		mPred.UnaryExpr<false>(*_cacheCompLL, CuLLIdxComputeFn(mLabels));
		_cacheCompMaxIdxs->BinaryExpr<false>(mLabels, *_cacheCompBinarized, CuLLIdxMaxEqFn());
	}
	// Vector mode
	else
	{
		mPred.BinaryExpr<false>(mLabels, *_cacheCompLL, CuLLVecComputeFn());
		mPred.BinaryExpr<false>(mLabels, *_cacheCompBinarized, CuLLVecMaxEqFn(*_cacheCompMaxIdxs));
	}

	logLoss = _cacheCompLL->Sum();

	numCorr = _cacheCompBinarized->Sum();

	CostMap ret;
	ret[CostMap::PRIMARY_NAME] = logLoss;
	ret["correct"] = numCorr;
	return ret;
}



Params CuLoglossCost::ComputeGrad(const Params& pred, const Params& labels)
{
	const CuMat &mPred = pred.GetCudaMatrix(_handle);
	const CuMat &mLabels = labels.GetCudaMatrix(_handle);

	_cacheCost->ResizeLike(mPred);
	CuMat *cost = new CuMat(*_cacheCost);

	if (_outputIsSoftmax)
	{
		if (labels.Rows == 1)
		{
			mPred.UnaryExpr<false>(*cost, CuLLIdxSoftmaxGradFn(mLabels));
		}
		else
		{
			mPred.BinaryExpr<false>(mLabels, *cost, CuScaledDiff(1.0f / pred.Cols));
			//AddScaled(mPred, 1.0f / pred.Cols, mLabels, -1.0f / pred.Cols, *cost);
		}
	}
	else
	{
		if (labels.Rows == 1)
		{
			mPred.UnaryExpr<false>(*cost, CuLLIdxGradFn(mLabels));
		}
		else
		{
			mPred.BinaryExpr<false>(mLabels, *cost, CuLLVecGradFn(mLabels));
		}
	}

	return Params(pred, cost);
}

void CuLoglossCost::SetOpIsSoftmax(bool value)
{
	_outputIsSoftmax = value;
}


