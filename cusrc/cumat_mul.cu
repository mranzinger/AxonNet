/*
 * File description: cumat_mul.cu
 * Author information: Mike Ranzinger mranzinger@alchemyapi.com
 * Copyright information: Copyright Orchestr8 LLC
 */

#include "cumat.cuh"

#include <hipblas.h>

#include <stdexcept>
#include <assert.h>

using namespace std;

CuMat operator*(const CuMat &a, const CuMat &b)
{
    return ScaledMultiply(1.0f, a, b);
}

CuMat operator*(const CuScopedWeakTranspose &tA, const CuMat &b)
{
    return ScaledMultiply(1.0f, tA, b);
}

CuMat operator*(const CuMat &a, const CuScopedWeakTranspose &tB)
{
    return ScaledMultiply(1.0f, a, tB);
}

CuMat operator*(const CuScopedWeakTranspose &tA, const CuScopedWeakTranspose &tB)
{
    return ScaledMultiply(1.0f, tA, tB);
}


CuMat ScaledMultiply(Real scale, const CuMat &a, const CuMat &b)
{
    CuMat dest(a._handle);

    ScaledMultiply(scale, a, b, 0.0f, dest);

    return dest;
}
CuMat ScaledMultiply(Real scale, const CuScopedWeakTranspose &tA, const CuMat &b)
{
    CuMat dest(b._handle);

    ScaledMultiply(scale, tA, b, 0.0f, dest);

    return dest;
}
CuMat ScaledMultiply(Real scale, const CuMat &a, const CuScopedWeakTranspose &tB)
{
    CuMat dest(a._handle);

    ScaledMultiply(scale, a, tB, 0.0f, dest);

    return dest;
}
CuMat ScaledMultiply(Real scale, const CuScopedWeakTranspose &tA, const CuScopedWeakTranspose &tB)
{
    CuMat dest(tA.Mat._handle);

    ScaledMultiply(scale, tA, tB, 0.0f, dest);

    return dest;
}

CuMat MultiplyTrans3D(const CuMat &a, uint32_t rows, uint32_t cols, const CuMat &b)
{
	CuMat dest(a._handle);

	MultiplyTrans3D(a, rows, cols, b, dest);

	return dest;
}

void ScaledMultiply(Real mulScale, const CuMat &a, const CuMat &b, Real scaleDest, CuMat &dest)
{
    assert(!a.Empty() && !b.Empty());
    assert(a._handle == b._handle);
    // Make sure the matrices are valid
    assert(a._cols == b._rows);

    // TODO: Support these other cases
    assert(a._storageOrder == CuColMajor && b._storageOrder == CuColMajor);

    if (scaleDest == 0.0f)
    {
        dest.Resize(a._rows, b._cols);
    }
    else
    {
        assert(dest._rows == a._rows && dest._cols == b._cols);
    }

    hipblasStatus_t status =
            hipblasSgemm(a._handle.CublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                            a._rows, b._cols, a._cols,
                            &mulScale, a._dMat, a._rows,
                            b._dMat, b._rows,
                            &scaleDest,
                            dest._dMat,
                            dest._rows);

    if (status != HIPBLAS_STATUS_SUCCESS)
        throw runtime_error("The matrix multiplication failed.");
}

void MultiplyTrans3D(const CuMat &a, uint32_t rows, uint32_t cols, const CuMat &b, CuMat &dest)
{
	assert(!a.Empty() && !b.Empty());
    assert(a._handle == b._handle);

    assert(a._storageOrder == CuColMajor && b._storageOrder == CuColMajor);

    assert(cols == b._rows);
    assert(a._rows == (rows * cols));

    dest.Resize(rows, b._cols);

    const float alpha = 1.0f, beta = 0.0f;

    for (uint32_t i = 0; i < b._cols; ++i)
    {
        hipblasStatus_t status =
                hipblasSgemm(a._handle.CublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                               rows, 1, cols,
                               &alpha,
                               a._dMat + i * a._rows,
                               rows,
                               b._dMat + i * b._rows,
                               b._rows,
                               &beta,
                               dest._dMat + i * rows,
                               rows);

        if (status != HIPBLAS_STATUS_SUCCESS)
            throw runtime_error("The matrix multiplication failed.");
    }

    /*const float *aBuffs[b._cols];
    const float *bBuffs[b._cols];
    float *cBuffs[b._cols];

    for (uint32_t i = 0; i < b._cols; ++i)
    {
    	aBuffs[i] = a._dMat + i * a._rows;
    	bBuffs[i] = b._dMat + i * b._rows;
    	cBuffs[i] = dest._dMat + i * rows;
    }

    hipblasStatus_t status =
    		hipblasSgemmBatched(a._handle.CublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    						   rows, 1, cols,
    						   &alpha,
    						   aBuffs, rows,
    						   bBuffs, b._rows,
    						   &beta,
    						   cBuffs, dest._rows,
    						   b._cols);

    if (status != HIPBLAS_STATUS_SUCCESS)
        throw runtime_error("The matrix multiplication failed.");*/
}

void ScaledMultiply(Real mulScale, const CuScopedWeakTranspose &tA,
                    const CuMat &b, Real scaleDest, CuMat &dest)
{
    const CuMat &a = tA.Mat;

    assert(a._rows == b._rows);
    assert(!a.Empty() && !b.Empty());
    assert(a._handle == b._handle);

    assert(a._storageOrder == CuColMajor && b._storageOrder == CuColMajor);

    if (scaleDest == 0.0f)
    {
        dest.Resize(a._cols, b._cols);
    }
    else
    {
        assert(dest._rows == a._cols && dest._cols == b._cols);
    }

    hipblasStatus_t status =
            hipblasSgemm(a._handle.CublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                           a._cols, b._cols, a._rows,
                           &mulScale, a._dMat, a._rows,
                           b._dMat, b._rows,
                           &scaleDest,
                           dest._dMat,
                           dest._rows);

    if (status != HIPBLAS_STATUS_SUCCESS)
        throw runtime_error("The matrix multiplication failed.");
}
void ScaledMultiply(Real mulScale, const CuMat &a,
                    const CuScopedWeakTranspose &tB, Real scaleDest, CuMat &dest)
{
    const CuMat &b = tB.Mat;

    assert(a._cols == b._cols);
    assert(!a.Empty() && !b.Empty());
    assert(a._handle == b._handle);

    assert(a._storageOrder == CuColMajor && b._storageOrder == CuColMajor);

    if (scaleDest == 0.0f)
    {
        dest.Resize(a._rows, b._rows);
    }
    else
    {
        assert(dest._rows == a._rows && dest._cols == b._rows);
    }

    hipblasStatus_t status =
            hipblasSgemm(a._handle.CublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                           a._rows, b._rows, a._cols,
                           &mulScale, a._dMat, a._rows,
                           b._dMat, b._rows,
                           &scaleDest,
                           dest._dMat,
                           dest._rows);

    if (status != HIPBLAS_STATUS_SUCCESS)
        throw runtime_error("The matrix multiplication failed.");
}



void ScaledMultiply(Real mulScale, const CuScopedWeakTranspose &tA,
                    const CuScopedWeakTranspose &tB, Real scaleDest, CuMat &dest)
{
    const CuMat &a = tA.Mat;
    const CuMat &b = tB.Mat;

    assert(a._rows == b._cols);
    assert(!a.Empty() && !b.Empty());
    assert(a._handle == b._handle);

    assert(a._storageOrder == CuColMajor && b._storageOrder == CuColMajor);

    if (scaleDest == 0.0f)
    {
        dest.Resize(a._cols, b._rows);
    }
    else
    {
        assert(dest._rows == a._cols && dest._cols == b._rows);
    }

    hipblasStatus_t status =
            hipblasSgemm(a._handle.CublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T,
                    a._cols, b._rows, a._rows,
                    &mulScale, a._dMat, a._rows,
                    b._dMat, b._rows,
                    &scaleDest,
                    dest._dMat,
                    dest._rows);

    if (status != HIPBLAS_STATUS_SUCCESS)
        throw runtime_error("The matrix multiplication failed.");
}




