#include "hip/hip_runtime.h"
/*
 * cu_convo_layer.cu
 *
 *  Created on: Jun 15, 2014
 *      Author: mike
 */

#include "cu_convo_layer.cuh"

#include "cusetup_provider.cuh"
#include "cu_weights.cuh"
#include "cumath_functions.cuh"

using namespace std;

class CuConvoLayer::Impl
{
private:
	CuContext _handle;

	CuWeights _weights;

	CuMat _cacheCompute;
	CuMat _cacheBackprop;
	CuMat _cacheWeightGrads;
	CuMat _cacheBiasGrads;
	Real **d_cacheWeightGrads,
	     **d_cacheBiasGrads;

	int _windowSizeX, _windowSizeY;
	int _padWidth, _padHeight;
	int _strideX, _strideY;

public:
	Impl(int deviceId,
		 int windowSizeX, int windowSizeY,
		 int strideX, int strideY,
		 int padWidth, int padHeight)
		: _windowSizeX(windowSizeX), _windowSizeY(windowSizeY),
		  _padWidth(padWidth), _padHeight(padHeight),
		  _strideX(strideX), _strideY(strideY),
		  d_cacheWeightGrads(NULL),
		  d_cacheBiasGrads(NULL)
	{
		_handle = CuSetupProvider::GetHandle(deviceId);

		_weights.SetHandle(_handle);

		_cacheCompute.SetHandle(_handle);
		_cacheCompute.SetSharedModify(true);

		_cacheBackprop.SetHandle(_handle);
		_cacheBackprop.SetSharedModify(true);

		_cacheWeightGrads.SetHandle(_handle);
		_cacheBiasGrads.SetHandle(_handle);
	}
	~Impl()
	{
	    hipFree(d_cacheWeightGrads);
	    hipFree(d_cacheBiasGrads);
	}

	Params Compute(const Params &input);
	Params Backprop(const Params &lastInput, const Params &lastOutput,
					const Params &outputErrors);

	void ComputeErrorGradient(const Params &lastInput, const Params &lastOutput,
	                          const Params &outputErrors);
	Params GetInputErrors(const Params &lastInput, const Params &lastOutput,
	                      const Params &outputErrors);

	void ApplyGradient();

	void SyncToDevice(const CWeights &hWeights, bool gradToo);
	void SyncToHost(CWeights &hWeights, bool gradToo) const;

	void SetLearningRate(Real rate);
	void SetMomentum(Real rate);
	void SetWeightDecay(Real rate);
	void InitCacheWeightGrads();
};

CuConvoLayer::CuConvoLayer(int deviceId,
						   int windowSizeX, int windowSizeY,
						   int strideX, int strideY,
						   int padWidth, int padHeight)
{
	_impl = new Impl(deviceId,
					 windowSizeX, windowSizeY,
					 strideX, strideY,
					 padWidth, padHeight);
}

CuConvoLayer::~CuConvoLayer()
{
	delete _impl;
}

Params CuConvoLayer::Compute(const Params& input) const
{
    return _impl->Compute(input);
}

Params CuConvoLayer::Backprop(const Params& lastInput,
		const Params& lastOutput, const Params& outputErrors)
{
    return _impl->Backprop(lastInput, lastOutput, outputErrors);
}

void CuConvoLayer::ApplyGradient()
{
    _impl->ApplyGradient();
}

void CuConvoLayer::SyncToDevice(const CWeights& hWeights, bool gradToo)
{
    _impl->SyncToDevice(hWeights, gradToo);
}

void CuConvoLayer::SyncToHost(CWeights& hWeights, bool gradToo) const
{
    _impl->SyncToHost(hWeights, gradToo);
}

void CuConvoLayer::SetLearningRate(Real rate)
{
    _impl->SetLearningRate(rate);
}

void CuConvoLayer::SetMomentum(Real rate)
{
    _impl->SetMomentum(rate);
}



void CuConvoLayer::SetWeightDecay(Real rate)
{
    _impl->SetWeightDecay(rate);
}

struct PlacementParams
{
    int KSkipStride;
    int KInnerSkipStride;
};

struct ConvoKernelParams
{
    int IpDepth;
    int OpDepth;
    int WindowSizeX;
    int WindowSizeY;
    int StrideX;
    int StrideY;
    int PadWidth;
    int PadHeight;

    Real *WeightsBuff;
    Real *BiasBuff;

    PlacementParams Places[20];
};

template<bool padded, int numImagesPerThread>
__global__ void CuConvoLayer_Compute(const Real *gInput, Real *gOutput,
                                     const Real *gWeights, const Real *gBiases,
                                     const int ipWidth, const int ipHeight, const int ipDepth,
                                     const int opWidth, const int opHeight, const int opDepth,
                                     const int wndSizeX, const int wndSizeY,
                                     const int strideX, const int strideY,
                                     const int padWidth, const int padHeight)
{
	__shared__ extern Real sInput[];

	// Switching the x's and the z's here
    const int destX = blockIdx.z * blockDim.z + threadIdx.z;

    const int destY = blockIdx.y * blockDim.y + threadIdx.y;

    const int layer = blockIdx.x * numImagesPerThread;

    const int dIdx = threadIdx.x;

    const int ipImgSize = ipWidth * ipHeight * ipDepth;

    const Real *lInput = gInput + layer * ipImgSize;

    const int opImgSize = opWidth * opHeight * opDepth;

    Real *lOutput = gOutput + layer * opImgSize;

    const int srcX = padded ? (-padWidth + destX * strideX) : (destX * strideX);
    const int srcY = padded ? (-padHeight + destY * strideY) : (destY * strideY);

    const int xMin = padded ? max(0, srcX) : srcX;
    const int yMin = padded ? max(0, srcY) : srcY;

    const int xMax = padded ? min(srcX + wndSizeX, ipWidth) : (srcX + wndSizeX);
    const int yMax = padded ? min(srcY + wndSizeY, ipHeight) : (srcY + wndSizeY);

    const int kSkipX = padded ? (xMin - srcX) : 0;
    const int kSkipY = padded ? (yMin - srcY) : 0;

    const int iStride = ipWidth * ipDepth;
    const int kStride = wndSizeX * ipDepth;

    const int kfSkipStride = padded ? ((kSkipY * kStride + kSkipX * ipDepth) * opDepth) : 0;
    const int kInnerSkipStride = padded ? ((kSkipX + (srcX + wndSizeX - xMax)) * ipDepth * opDepth) : 0;

    const int xEnd = xMax * ipDepth;

    const int dxMin = xMin * ipDepth;

    //int imgIdx = yMin * iStride;
    int weightsIdx = padded ? (dIdx + kfSkipStride) : dIdx;

    const int endImgIdx = yMax * iStride;

    const int procInputWidth = padded ? (xEnd - dxMin) : (wndSizeX * ipDepth);
    const int procInputSize = padded ? (procInputWidth * (yMax - yMin)) : (procInputWidth * wndSizeY);

    /// !!!! Load the image buffer into shared memory !!!!
    // Calculate the number of warps that are in this block.
    // For coalesced access rules, we want these guys to be grouped on a row
    const int numWarps = blockDim.x / 32;

    // Not enough threads to even fill a single warp...
    // This will not be ultra-efficient
    if (numWarps <= 1)
    {
        const int startCol = dxMin + threadIdx.x;

        for (int iY = 0, imgIdx = yMin * iStride;
                imgIdx < endImgIdx;
                ++iY, imgIdx += iStride)
        {
            for (int iX = startCol; iX < xEnd; iX += blockDim.x)
            {
				#pragma unroll
            	for (int k = 0; k < numImagesPerThread; ++k)
            	{
            		const Real iVal = lInput[imgIdx + (k * ipImgSize) + iX];

            		sInput[(k * procInputSize) + (iY * procInputWidth + iX - dxMin)] = iVal;
            	}
            }
        }
    }
    else
    {
        const int warpsPerRow = round_up(numWarps, yMax - yMin);
        const int simulRows = numWarps / warpsPerRow;

        // Let each warp do a separate row
        const int startRow = threadIdx.x / (32 * warpsPerRow);
        const int startCol = dxMin + (threadIdx.x % (32 * warpsPerRow));

        for (int iY = startRow, imgIdx = (yMin + startRow) * iStride;
                 imgIdx < endImgIdx;
                 iY += simulRows, imgIdx += (simulRows * iStride))
        {
            for (int iX = startCol; iX < xEnd; iX += (32 * warpsPerRow))
            {
				#pragma unroll
            	for (int k = 0; k < numImagesPerThread; ++k)
            	{
            		const Real iVal = lInput[imgIdx + (k * ipImgSize) + iX];

            		sInput[(k * procInputSize) + (iY * procInputWidth + iX - dxMin)] = iVal;
            	}
            }
        }
    }

    __syncthreads();

    //Real sum = gBiases[dIdx];
    Real sum[numImagesPerThread] = { 0.0f };

    // Peel vectors of 8
    const int vecProcX = procInputWidth & ~0x7;
    const int vecTailX = procInputWidth & 0x7;

    const int vecXend = dxMin + vecProcX;

    int ipIdx = 0;
    for (int iY = yMin; iY < yMax; ++iY)
    {
    	for (int iX = dxMin; iX < vecXend; iX += 8)
    	{
			#pragma unroll
    		for (int i = 0; i < 8; ++i)
    		{
    			const Real kVal = gWeights[weightsIdx + i * opDepth];

				#pragma unroll
    			for (int k = 0; k < numImagesPerThread; ++k)
    			{
    				const Real iVal = sInput[ipIdx + (k * procInputSize) + i];

    				const Real product = iVal * kVal;

    				sum[k] += product;
    			}
    		}

    		ipIdx += 8;
    		weightsIdx += 8 * opDepth;
    	}

#define DUFF_CASE(v) case v: \
			{ \
			const Real kVal = gWeights[weightsIdx + (v - 1) * opDepth]; \
			_Pragma("unroll") \
    		for (int k = 0; k < numImagesPerThread; ++k) \
    		{ \
    			sum[k] += sInput[ipIdx + (k * procInputSize) + (v - 1)] * kVal; \
    		} }

    	switch (vecTailX)
    	{
    	DUFF_CASE(7);
    	DUFF_CASE(6);
    	DUFF_CASE(5);
    	DUFF_CASE(4);
    	DUFF_CASE(3);
    	DUFF_CASE(2);
    	DUFF_CASE(1);
    	case 0:
    		break;
    	}

    	ipIdx += vecTailX;
    	weightsIdx += (padded ? kInnerSkipStride : 0) + vecTailX * opDepth;

#undef DUFF_CASE

        // Skip over the padding parts of the filter
        //weightsIdx += kInnerSkipStride;
    }

    const int opStoreIdx = destY * opWidth * opDepth + destX * opDepth;

    // Finally, store the sum
    //lOutput[opStoreIdx + dIdx] = sum;
    const Real bias = gBiases[dIdx];
	#pragma unroll
    for (int k = 0; k < numImagesPerThread; ++k)
    {
    	lOutput[opStoreIdx + (k * opImgSize) + dIdx] = sum[k] + bias;
    }
}

Params CuConvoLayer::Impl::Compute(const Params& input)
{
    const CuMat &mInput = input.GetCudaMatrix(_handle);

    const int ipWidth = input.Width;
    const int ipHeight = input.Height;
	const int ipDepth = input.Depth;
	const int batchSize = input.Cols;

	const int ipEffectiveWidth = ipWidth + _padWidth * 2,
		      ipEffectiveHeight = ipHeight + _padHeight * 2;

	const int opWidth = (int) floor((ipEffectiveWidth - _windowSizeX) / float(_strideX)) + 1;
	const int opHeight = (int) floor((ipEffectiveHeight - _windowSizeY) / float(_strideY)) + 1;
	const int opDepth = _weights.Weights.Rows();

	_cacheCompute.Resize(opWidth * opHeight * opDepth, batchSize);
	Params output(opWidth, opHeight, opDepth,
	            new CuMat(_cacheCompute));

	CuMat &mOutput = output.GetCudaMatrix(_handle);

	hipError_t err = hipSetDevice(_handle.Device);

	if (err != hipSuccess)
	    throw runtime_error("Unable to set the device.");

	if (opDepth > 1024)
		throw runtime_error("Output depths greater than 1024 are not supported.");

	uint32_t blockDepth = opDepth;

	dim3 blockSize(blockDepth, 1, 1);
	dim3 gridSize = round_up(blockDepth * batchSize, opHeight, opWidth, blockSize);

	uint32_t smemSize = _windowSizeX * _windowSizeY * ipDepth * sizeof(Real);

	uint32_t numImagesPerThread = 1;
	for (int i = 4; i > 1; --i)
	{
		if ((batchSize % i) == 0)
		{
			numImagesPerThread = i;
			break;
		}
	}

	smemSize *= numImagesPerThread;
	gridSize.x /= numImagesPerThread;

	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(CuConvoLayer_Compute), hipFuncCachePreferShared);

	bool padded = _padWidth > 0 || _padHeight > 0;

#define LAUNCH_CONVO_KERNEL(p, v) \
			CuConvoLayer_Compute \
				<p, v> \
				<<<gridSize, blockSize, smemSize>>> \
					(mInput.Buff(), mOutput.Buff(), \
				     _weights.Weights.Buff(), _weights.Biases.Buff(), \
				     ipWidth, ipHeight, ipDepth, \
				     opWidth, opHeight, opDepth, \
				     _windowSizeX, _windowSizeY, \
				     _strideX, _strideY, \
				     _padWidth, _padHeight)

#define PADDED_B(v) \
	if (padded) \
		LAUNCH_CONVO_KERNEL(true, v); \
	else \
		LAUNCH_CONVO_KERNEL(false, v)

	switch (numImagesPerThread)
	{
	case 1:
		PADDED_B(1);
		break;
	case 2:
		PADDED_B(2);
		break;
	case 3:
		PADDED_B(3);
		break;
	case 4:
		PADDED_B(4);
		break;
	}

#undef PADDED_B
#undef LAUNCH_CONVO_KERNEL

	/*CuConvoLayer_Compute
        <<<gridSize, blockSize, smemSize>>>
                        (mInput.Buff(), mOutput.Buff(),
                         _weights.Weights.Buff(), _weights.Biases.Buff(),
                         ipWidth, ipHeight, ipDepth,
                         opWidth, opHeight, opDepth,
                         _windowSizeX, _windowSizeY,
                         _strideX, _strideY,
                         _padWidth, _padHeight);*/

	err = hipGetLastError();

	if (err != hipSuccess)
		throw runtime_error("Unable to compute convolution.");

	return output;


}

template<bool padded, int numImagesPerThread>
__global__ void CuConvoLayer_NaiveBackprop(const Real *gOutputErrors, Real *gInputErrors,
										   const Real *gWeights,
										   const int ipWidth, const int ipHeight, const int ipDepth,
										   const int opWidth, const int opHeight, const int opDepth,
										   const int wndSizeX, const int wndSizeY,
										   const int strideX, const int strideY,
										   const int padWidth, const int padHeight)
{
	__shared__ extern Real shared_module[];

	const int destX = blockIdx.y * blockDim.y + threadIdx.y;
	const int destY = blockIdx.z * blockDim.z + threadIdx.z;

	const int layer = blockIdx.x * numImagesPerThread;

	const int ipImgStride = ipWidth * ipDepth;
	const int ipImgSize = ipImgStride * ipHeight;
	const int opImgSize = opWidth * opHeight * opDepth;

	const Real *lOutputErrors = gOutputErrors + opImgSize * layer;
	Real *lInputErrors = gInputErrors + ipImgSize * layer;

	const int ipModuleSize = wndSizeX * wndSizeY * ipDepth;

	// Compute the input error module.
	// No need to worry about padding here
	{
		const int weightsSize = ipModuleSize * opDepth;

		// The weights matrix is column major, which means that each thread
		// will operate on contiguous memory.
		const int startIdx = threadIdx.x * opDepth;
		const int threadStride = blockDim.x * opDepth;

		const int opErrIdx = destY * opWidth * opDepth + destX * opDepth;

		// A thread block doesn't necessarily process the entire block
		// at once
		for (int currRow = startIdx, i = threadIdx.x; currRow < weightsSize;
				currRow += threadStride, i += blockDim.x)
		{
			//Real val = 0.0f;
		    //Real vals[numImagesPerThread] = { 0 };

            #pragma unroll
		    for (int k = 0; k < numImagesPerThread; ++k)
		    {
		        shared_module[k * ipModuleSize + i] = 0.0f;
		    }

			for (int wI = 0; wI < opDepth; ++wI)
			{
				const Real wVal = gWeights[currRow + wI];

                #pragma unroll
				for (int k = 0; k < numImagesPerThread; ++k)
				{
				    const Real errVal = lOutputErrors[k * opImgSize + opErrIdx + wI];

				    shared_module[(k * ipModuleSize) + i] += wVal * errVal;
				}
			}

			//shared_module[i] = val;
		}

		// Ok, at this point, all of the input errors for this module are stored in
		// shared memory. The next step is to write this module out into the input
		// error buffer.
		__syncthreads();
	}

	// Be lazy about padding...
	const int srcX = padded ? (-padWidth + destX * strideX) : (destX * strideX);
    const int srcY = padded ? (-padHeight + destY * strideY) : (destY * strideY);

    // We know that the thread block size is a factor of the module stride
    const int yStart = max(srcY, 0);
    const int yEnd = min(srcY + wndSizeY, ipHeight);

    const int xStart = max(srcX * ipDepth, 0);
    const int xEnd = min((srcX + wndSizeX), ipWidth) * ipDepth;

    const int yOff = max(-srcY, 0);
    const int xOff = max(-srcX, 0) * ipDepth;

    const int moduleStride = wndSizeX * ipDepth;

    int opYIdx = yStart * ipImgStride;
    int ipYIdx = yOff * moduleStride;
    for (int y = yStart; y < yEnd; ++y, opYIdx += ipImgStride, ipYIdx += moduleStride)
    {
    	for (int opX = xStart + threadIdx.x, ipX = xOff + threadIdx.x; opX < xEnd; opX += blockDim.x, ipX += blockDim.x)
    	{
            #pragma unroll
    	    for (int k = 0; k < numImagesPerThread; ++k)
    	    {
    	        const Real sVal = shared_module[(k * ipModuleSize) + ipYIdx + ipX];

    	        Real *dVal = lInputErrors + (k * ipImgSize) + opYIdx + opX;

    	        // TODO: It is really ugly to use atomics here...
    	        atomicAdd(dVal, sVal);
    	    }
    	}
    }
}

Params CuConvoLayer::Impl::Backprop(const Params& lastInput,
        const Params& lastOutput, const Params& outputErrors)
{
	Params ret = GetInputErrors(lastInput, lastOutput, outputErrors);

	ComputeErrorGradient(lastInput, lastOutput, outputErrors);

	return ret;
}

Params CuConvoLayer::Impl::GetInputErrors(const Params& lastInput, const Params& lastOutput,
        const Params& outputErrors)
{
    const int ipWidth = lastInput.Width;
    const int ipHeight = lastInput.Height;
    const int ipDepth = lastInput.Depth;
    const int batchSize = lastInput.Cols;

    const int opWidth = lastOutput.Width;
    const int opHeight = lastOutput.Height;
    const int opDepth = lastOutput.Depth;

    _cacheBackprop.Resize(ipWidth * ipHeight * ipDepth, batchSize);
    Params inputErrors(ipWidth, ipHeight, ipDepth,
               new CuMat(_cacheBackprop));

    const CuMat &mLastInput = lastInput.GetCudaMatrix(_handle);
    const CuMat &mOutputErrors = outputErrors.GetCudaMatrix(_handle);
    CuMat &mInputErrors = inputErrors.GetCudaMatrix(_handle);

    hipStreamSynchronize(mOutputErrors.Handle().GetStream());

    // Initialize the input error matrix to 0
    mInputErrors.SetConstant(0.0f);

    hipError_t err = hipSetDevice(_handle.Device);

    if (err != hipSuccess)
        throw runtime_error("Unable to set the device.");

    if (opDepth > 1024)
        throw runtime_error("Output depths greater than 1024 are not supported.");

    //if ((opDepth % 32) != 0)
    //  throw runtime_error("Only output depths that have 32 as a factor are currently supported.");

    uint32_t moduleSize = _windowSizeX * _windowSizeY * ipDepth;

    uint32_t patchSeg = _windowSizeX * ipDepth;
    if (patchSeg > 1024)
        patchSeg = max(_windowSizeX, ipDepth);
    if (patchSeg > 1024)
        patchSeg = min(_windowSizeX, ipDepth);

    assert(patchSeg <= 1024);

    // Similar to compute, the x dimension will be used as the z dimension
    dim3 blockSize(patchSeg, 1, 1);
    dim3 gridSize = round_up(batchSize * patchSeg, opWidth, opHeight, blockSize);

    uint32_t smemSize = moduleSize * sizeof(Real);

    uint32_t numImagesPerThread = 1;
    for (int i = 4; i > 1; --i)
    {
        if ((batchSize % i) == 0)
        {
            numImagesPerThread = i;
            break;
        }
    }

    smemSize *= numImagesPerThread;
    gridSize.x /= numImagesPerThread;

    bool padded = _padWidth > 0 || _padHeight > 0;


    // The BP kernel computes the input errors
#define LAUNCH_BP_KERNEL(p, v) \
            CuConvoLayer_NaiveBackprop \
                <p, v> \
                <<<gridSize, blockSize, smemSize>>> \
                    (mOutputErrors.Buff(), mInputErrors.Buff(), \
                     _weights.Weights.Buff(), \
                     ipWidth, ipHeight, ipDepth, \
                     opWidth, opHeight, opDepth, \
                     _windowSizeX, _windowSizeY, \
                     _strideX, _strideY, \
                     _padWidth, _padHeight)

#define PADDED_B(v) \
    if (padded) \
        LAUNCH_BP_KERNEL(true, v); \
    else \
        LAUNCH_BP_KERNEL(false, v)

    switch (numImagesPerThread)
    {
    case 1:
        PADDED_B(1);
        break;
    case 2:
        PADDED_B(2);
        break;
    case 3:
        PADDED_B(3);
        break;
    case 4:
        PADDED_B(4);
        break;
    }

#undef PADDED_B
#undef LAUNCH_BP_KERNEL

    return inputErrors;
}

__device__ uint32_t get_smid(void)
{
     uint32_t ret;

     asm("mov.u32 %0, %smid;" : "=r"(ret) );

     return ret;
}

template<bool padded, int numImagesPerThread>
__global__ void CuConvoLayer_ComputeWeightGrad(
                        const Real *gOutputErrors, const Real *gLastInput,
                        Real **gWeightsGrad, Real **gBiasGrad,
                        const int ipWidth, const int ipHeight, const int ipDepth,
                        const int opWidth, const int opHeight, const int opDepth,
                        const int wndSizeX, const int wndSizeY,
                        const int strideX, const int strideY,
                        const int padWidth, const int padHeight)
{
    __shared__ extern Real shared_input[];

    // Switching the x's and the z's here
    const int destX = blockIdx.y * blockDim.y + threadIdx.y;
    const int destY = blockIdx.z * blockDim.z + threadIdx.z;

    const int layer = blockIdx.x * numImagesPerThread;

    const int dIdx = threadIdx.x;

    const int ipImgSize = ipWidth * ipHeight * ipDepth;

    const Real *lLastInput = gLastInput + layer * ipImgSize;

    const int opImgStride = opWidth * opDepth;
    const int opImgSize = opImgStride * opHeight;

    const Real *lOutputErrors = gOutputErrors + layer * opImgSize;

    const int srcX = padded ? (-padWidth + destX * strideX) : (destX * strideX);
    const int srcY = padded ? (-padHeight + destY * strideY) : (destY * strideY);

    const int xMin = padded ? max(0, srcX) : srcX;
    const int yMin = padded ? max(0, srcY) : srcY;

    const int xMax = padded ? min(srcX + wndSizeX, ipWidth) : (srcX + wndSizeX);
    const int yMax = padded ? min(srcY + wndSizeY, ipHeight) : (srcY + wndSizeY);

    const int kSkipX = padded ? (xMin - srcX) : 0;
    const int kSkipY = padded ? (yMin - srcY) : 0;

    const int iStride = ipWidth * ipDepth;
    const int kStride = wndSizeX * ipDepth;

    const int kfSkipStride = padded ? ((kSkipY * kStride + kSkipX * ipDepth) * opDepth) : 0;
    const int kInnerSkipStride = padded ? ((kSkipX + (srcX + wndSizeX - xMax)) * ipDepth * opDepth) : 0;

    const int xEnd = xMax * ipDepth;

    const int dxMin = xMin * ipDepth;

    const int endImgIdx = yMax * iStride;

    const int procInputWidth = padded ? (xEnd - dxMin) : (wndSizeX * ipDepth);
    const int procInputSize = padded ? (procInputWidth * (yMax - yMin)) : (procInputWidth * wndSizeY);

    /// !!!! Load the image buffer into shared memory !!!!
    // Calculate the number of warps that are in this block.
    // For coalesced access rules, we want these guys to be grouped on a row
    const int numWarps = blockDim.x / 32;

    // Not enough threads to even fill a single warp...
    // This will not be ultra-efficient
    if (numWarps <= 1)
    {
        const int startCol = dxMin + threadIdx.x;

        for (int iY = 0, imgIdx = yMin * iStride;
                imgIdx < endImgIdx;
                ++iY, imgIdx += iStride)
        {
            for (int iX = startCol; iX < xEnd; iX += blockDim.x)
            {
                #pragma unroll
                for (int k = 0; k < numImagesPerThread; ++k)
                {
                    const Real iVal = lLastInput[imgIdx + (k * ipImgSize) + iX];

                    shared_input[(k * procInputSize) + (iY * procInputWidth + iX - dxMin)] = iVal;
                }
            }
        }
    }
    else
    {
        const int warpsPerRow = round_up(numWarps, yMax - yMin);
        const int simulRows = numWarps / warpsPerRow;

        // Let each warp do a separate row
        const int startRow = threadIdx.x / (32 * warpsPerRow);
        const int startCol = dxMin + (threadIdx.x % (32 * warpsPerRow));

        for (int iY = startRow, imgIdx = (yMin + startRow) * iStride;
                 imgIdx < endImgIdx;
                 iY += simulRows, imgIdx += (simulRows * iStride))
        {
            for (int iX = startCol; iX < xEnd; iX += (32 * warpsPerRow))
            {
                #pragma unroll
                for (int k = 0; k < numImagesPerThread; ++k)
                {
                    const Real iVal = lLastInput[imgIdx + (k * ipImgSize) + iX];

                    shared_input[(k * procInputSize) + (iY * procInputWidth + iX - dxMin)] = iVal;
                }
            }
        }
    }

    __syncthreads();

    // Now that the input buffer has been loaded, we need to write into the weight
    // gradient buffer.
    // The simple form of the equation is:
    // W_g = err * input^T

    // We will use the current streaming multiprocessor as the destination buffer.
    // This should help reduce atomic conflict
    const uint32_t smIdx = get_smid();
    //const uint32_t smIdx = 0;

    Real *lWeightsGrad = gWeightsGrad[smIdx];
    Real *lBiasGrad = gBiasGrad[smIdx];

    // Each thread will process a row of this outer product. The upshot of this
    // is that we can keep the output error value cached as a local value.
    // The input values are also in shared memory, so the expensive part of this
    // operation is writing into the gradient buffer
    Real opErrVals[numImagesPerThread];
    #pragma unroll
    for (int i = 0; i < numImagesPerThread; ++i)
    {
        opErrVals[i] = lOutputErrors[i * opImgSize       // Index the output error image
                                  + destY * opImgStride  // Index the row of the output image
                                  + destX * opDepth      // Index the column block of the output image
                                  + threadIdx.x];        // Index the output error value
    }

    // Peel vectors of 8
    const int vecProcX = procInputWidth & ~0x7;
    const int vecTailX = procInputWidth & 0x7;

    const int vecXend = dxMin + vecProcX;

    int weightsIdx = padded ? (dIdx + kfSkipStride) : dIdx;

    int ipIdx = 0;
    for (int iY = yMin; iY < yMax; ++iY)
    {
        for (int iX = dxMin; iX < vecXend; iX += 8)
        {
            #pragma unroll
            for (int i = 0; i < 8; ++i)
            {
                // We can do a pre-sum of the various images before writing
                // it into the gradient buffer, which in effect cuts the number
                // of atomic operations proportionally to the number of images
                // per thread
                Real val = 0.0f;

                #pragma unroll
                for (int k = 0; k < numImagesPerThread; ++k)
                {
                    const Real iVal = shared_input[ipIdx + (k * procInputSize) + i];

                    val += opErrVals[k] * iVal;
                }

                Real *pWeight = lWeightsGrad + weightsIdx + i * opDepth;

                atomicAdd(pWeight, val);
            }

            ipIdx += 8;
            weightsIdx += 8 * opDepth;
        }

        for (int v = 0; v < vecTailX; ++v)
        {
            Real val = 0.0f;

            #pragma unroll
            for (int k = 0; k < numImagesPerThread; ++k)
            {
                const Real iVal = shared_input[ipIdx + (k * procInputSize) + v];

                val += opErrVals[k] * iVal;
            }

            Real *pWeight = lWeightsGrad + weightsIdx + v * opDepth;

            atomicAdd(pWeight, val);
        }

        ipIdx += vecTailX;
        weightsIdx += (padded ? kInnerSkipStride : 0) + vecTailX * opDepth;
    }

    // Don't forget about the biases!
    #pragma unroll
    for (int k = 0; k < numImagesPerThread; ++k)
    {
        Real *pBiasDest = lBiasGrad + dIdx;

        atomicAdd(pBiasDest, opErrVals[k]);
    }
}

template<uint32_t NumBuffs>
__global__ void CuConvoLayer_SumGradients(Real **grads,
                                          Real *dest,
                                          uint32_t gradBuffSize
                                          )
{
    const uint32_t tIdx = blockDim.x * blockIdx.x + threadIdx.x;

    if (tIdx >= gradBuffSize)
        return;

    Real sum = 0.0f;

    #pragma unroll
    for (uint32_t k = 0; k < NumBuffs; ++k)
    {
        const Real val = grads[k][tIdx];

        sum += val;
    }

    dest[tIdx] = sum;
}

void CuConvoLayer::Impl::ComputeErrorGradient(const Params& lastInput,
                                              const Params& lastOutput,
                                              const Params& outputErrors)
{
    const int ipWidth = lastInput.Width;
    const int ipHeight = lastInput.Height;
    const int ipDepth = lastInput.Depth;
    const int batchSize = lastInput.Cols;

    const int opWidth = lastOutput.Width;
    const int opHeight = lastOutput.Height;
    const int opDepth = lastOutput.Depth;

    hipError_t err = hipSetDevice(_handle.Device);

    if (err != hipSuccess)
        throw runtime_error("Unable to set the device.");

    InitCacheWeightGrads();

    const CuMat &mLastInput = lastInput.GetCudaMatrix(_handle);
    const CuMat &mOutputErrors = outputErrors.GetCudaMatrix(_handle);

    // Set all of the gradients to zero
    _cacheWeightGrads.SetConstant(0.0f);
    _cacheBiasGrads.SetConstant(0.0f);
    _weights.WeightsGrad.SetConstant(0.0f);
    _weights.BiasGrad.SetConstant(0.0f);

    if (opDepth > 1024)
        throw runtime_error("Output depths greater than 1024 are not supported.");

    uint32_t blockDepth = opDepth;

    dim3 blockSize(blockDepth, 1, 1);
    dim3 gridSize = round_up(blockDepth * batchSize, opWidth, opHeight, blockSize);

    uint32_t smemSize = _windowSizeX * _windowSizeY * ipDepth * sizeof(Real);

    uint32_t numImagesPerThread = 1;
    for (int i = 4; i > 1; --i)
    {
        if ((batchSize % i) == 0)
        {
            numImagesPerThread = i;
            break;
        }
    }

    smemSize *= numImagesPerThread;
    gridSize.x /= numImagesPerThread;

    bool padded = _padWidth > 0 || _padHeight > 0;

#define LAUNCH_BP_KERNEL(p, v) \
    CuConvoLayer_ComputeWeightGrad \
        <p, v> \
        <<<gridSize, blockSize, smemSize>>> \
            (mOutputErrors.Buff(), mLastInput.Buff(), \
             d_cacheWeightGrads, d_cacheBiasGrads, \
             ipWidth, ipHeight, ipDepth, \
             opWidth, opHeight, opDepth, \
             _windowSizeX, _windowSizeY, \
             _strideX, _strideY, \
             _padWidth, _padHeight)

#define PADDED_B(v) \
    if (padded) \
        LAUNCH_BP_KERNEL(true, v); \
    else \
        LAUNCH_BP_KERNEL(false, v)

    switch (numImagesPerThread)
    {
    case 1:
        PADDED_B(1);
        break;
    case 2:
        PADDED_B(2);
        break;
    case 3:
        PADDED_B(3);
        break;
    case 4:
        PADDED_B(4);
        break;
    }

#undef PADDED_B
#undef LAUNCH_BP_KERNEL

    blockSize = dim3(min(1024, _weights.WeightsGrad.Size()), 1, 1);
    gridSize = round_up(_weights.WeightsGrad.Size(), 1, 1, blockSize);

    // Sum all of the partial weights buffers
    CuConvoLayer_SumGradients
        <16>
        <<<gridSize, blockSize>>>
            (d_cacheWeightGrads,
             _weights.WeightsGrad.Buff(),
             _weights.WeightsGrad.Size()
             );

    blockSize = dim3(min(1024, _weights.BiasGrad.Size()), 1, 1);
    gridSize = round_up(_weights.BiasGrad.Size(), 1, 1, blockSize);

    CuConvoLayer_SumGradients
        <16>
        <<<gridSize, blockSize>>>
            (d_cacheBiasGrads,
             _weights.BiasGrad.Buff(),
             _weights.BiasGrad.Size()
             );

    _weights.DynamicLearningRate = 1.f / (opWidth * opHeight);
}

void CuConvoLayer::Impl::ApplyGradient()
{
    _weights.ApplyGradient();
}

void CuConvoLayer::Impl::SyncToDevice(const CWeights& hWeights, bool gradToo)
{
    _weights.CopyToDevice(hWeights, gradToo);
}

void CuConvoLayer::Impl::SyncToHost(CWeights& hWeights, bool gradToo) const
{
    _weights.CopyToHost(hWeights, gradToo);
}

void CuConvoLayer::Impl::SetLearningRate(Real rate)
{
    _weights.LearningRate = rate;
}

void CuConvoLayer::Impl::SetMomentum(Real rate)
{
    _weights.Momentum = rate;
}

void CuConvoLayer::Impl::SetWeightDecay(Real rate)
{
    _weights.WeightDecay = rate;
}

void CuConvoLayer::Impl::InitCacheWeightGrads()
{
    if (d_cacheWeightGrads)
        return;

    static const uint32_t s_numSMS = 16;

    // Each SM will get it's own weights gradient buffer. It would be much smarter if layers shared
    // buffers like this. Two convo layers don't need different buffers
    _cacheWeightGrads.Resize(_weights.WeightsGrad.Rows(), _weights.WeightsGrad.Cols() * s_numSMS);
    _cacheBiasGrads.Resize(_weights.BiasGrad.Rows(), _weights.BiasGrad.Cols() * s_numSMS);

    hipMalloc(&d_cacheWeightGrads, sizeof(Real*) * s_numSMS);
    hipMalloc(&d_cacheBiasGrads, sizeof(Real*) * s_numSMS);

    Real *dWeightBuff = _cacheWeightGrads.Buff();
    Real *dBiasBuff = _cacheBiasGrads.Buff();

    Real *hWeightBuffs[s_numSMS],
         *hBiasBuffs[s_numSMS];
    for (uint32_t i = 0; i < s_numSMS; ++i)
    {
        hWeightBuffs[i] = dWeightBuff + (i * _weights.WeightsGrad.Size());
        hBiasBuffs[i] = dBiasBuff + (i * _weights.BiasGrad.Size());
    }

    // Copy the gradient buffer pointers to the device
    hipMemcpy(d_cacheWeightGrads, hWeightBuffs, sizeof(hWeightBuffs), hipMemcpyHostToDevice);
    hipMemcpy(d_cacheBiasGrads, hBiasBuffs, sizeof(hBiasBuffs), hipMemcpyHostToDevice);
}







































