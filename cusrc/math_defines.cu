
#include "math_defines.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

hipStream_t CuContext::GetStream() const
{
    hipStream_t ret;
    hipblasGetStream(CublasHandle, &ret);
    return ret;
}
void CuContext::SetStream(hipStream_t stream)
{
    hipblasSetStream(CublasHandle, stream);
}
