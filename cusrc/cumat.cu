#include "cumat.cuh"

#include <hipblas.h>

#include <stdexcept>
#include <assert.h>

using namespace std;

CuMat::CuMat()
	: _handle(0), _refCt(NULL), _dMat(NULL), _rows(0), _cols(0), _storageOrder(CuColMajor)
{
	_refCt = new int(1);
}

CuMat::CuMat(hipblasHandle_t handle,
		     unsigned long rows, unsigned long cols, 
		     CuStorageOrder order)
	: _handle(handle), _dMat(NULL), _rows(rows), _cols(cols), _storageOrder(order)
{
	_refCt = new int(1);
	
	hipError_t cudaStat = hipMalloc(&_dMat, rows * cols * sizeof(Real));
	if (cudaStat != hipSuccess)
		throw runtime_error("Unable to allocate the specified matrix");
}

CuMat::CuMat(const CuMat &other)
	: _handle(other._handle), _dMat(other._dMat), _rows(other._rows), _cols(other._cols),
	  _refCt(other._refCt), _storageOrder(other._storageOrder)
{
	// Increment the ref count
	++(*_refCt);
}

CuMat::~CuMat()
{
	// Decrement the ref count
	--(*_refCt);
	if (*_refCt == 0)
	{
		delete _refCt;
		// Free the device memory
		hipFree(_dMat);
	}
}

CuMat &CuMat::operator=(CuMat other)
{
	swap(*this, other);
	return *this;
}

CuMat CuMat::Copy() const
{
	CuMat ret(_handle, _rows, _cols, _storageOrder);
	
	if (_dMat)
	{
		hipError_t status = hipMemcpy(ret._dMat, _dMat, _rows * _cols * sizeof(Real),
					   hipMemcpyDeviceToDevice);
		
		if (status != hipSuccess)
			throw runtime_error("Unable to copy the device memory from this matrix into the copy.");
	}
	
	return ret;
}

void CuMat::CopyToDevice(const Real *hMatrix)
{
	hipblasStatus_t status = hipblasSetMatrix(_rows, _cols, sizeof(Real),
											hMatrix, _rows, _dMat, _rows);
	
	if (status != HIPBLAS_STATUS_SUCCESS)
		throw runtime_error("Unable to copy the host matrix to the device.");
}

void CuMat::CopyToDevice(const CMatrix &hMatrix)
{
	assert(_rows == hMatrix.rows() &&
		   _cols == hMatrix.cols());
	
	CopyToDevice(hMatrix.data());
}

void CuMat::CopyToDevice(const RMatrix &hMatrix)
{
	CMatrix cMat = hMatrix;
	CopyToDevice(cMat);
}

void CuMat::CopyToDeviceAsync(const Real *hMatrix, hipStream_t stream)
{
	hipblasStatus_t status = hipblasSetMatrixAsync(_rows, _cols, sizeof(Real),
								hMatrix, _rows, _dMat, _rows, stream);
	
	if (status != HIPBLAS_STATUS_SUCCESS)
		throw runtime_error("Unable to copy the host matrix to the device.");
}

void CuMat::CopyToDeviceAsync(const CMatrix &hMatrix, hipStream_t stream)
{
	assert(_rows == hMatrix.rows() &&
		   _cols == hMatrix.cols());
	
	CopyToDeviceAsync(hMatrix.data(), stream);
}

void CuMat::CopyToDeviceAsync(const RMatrix &hMatrix, hipStream_t stream)
{
	CMatrix cMat = hMatrix;
	CopyToDeviceAsync(cMat, stream);
}

void CuMat::CopyToHost(Real* hMatrix) const
{
	hipblasStatus_t status = hipblasGetMatrix(_rows, _cols, sizeof(Real),
											_dMat, _rows, hMatrix, _rows);

	if (status != HIPBLAS_STATUS_SUCCESS)
		throw runtime_error("Unable to copy the device matrix to the host.");
}

void CuMat::CopyToHost(CMatrix& hMatrix) const
{
	CopyToHost(hMatrix.data());
}

void CuMat::CopyToHost(RMatrix& hMatrix) const
{
	CMatrix cMat(hMatrix.rows(), hMatrix.cols());
	CopyToHost(cMat);
	hMatrix = cMat;
}

void CuMat::CopyToHostAsync(Real* hMatrix, hipStream_t stream)
{
	hipblasStatus_t status = hipblasGetMatrixAsync(_rows, _cols, sizeof(Real),
								_dMat, _rows, hMatrix, _rows, stream);

	if (status != HIPBLAS_STATUS_SUCCESS)
		throw runtime_error("Unable to copy the device matrix to the host.");
}

void CuMat::CopyToHostAsync(CMatrix& hMatrix, hipStream_t stream)
{
	CopyToHostAsync(hMatrix.data(), stream);
}

void swap(CuMat &a, CuMat &b)
{
	swap(a._handle, b._handle);
	swap(a._dMat, b._dMat);
	swap(a._rows, b._rows);
	swap(a._cols, b._cols);
	swap(a._refCt, b._refCt);
	swap(a._storageOrder, b._storageOrder);
}
