#include "cumat.cuh"

#include <hipblas.h>

#include <stdexcept>
#include <assert.h>

using namespace std;

CuMat::CuMat()
	: _handle(0), _dMat(NULL), _rows(0), _cols(0), _storageOrder(CuColMajor)
{
	_refCt = new uint32_t(1);
}

CuMat::CuMat(hipblasHandle_t handle,
		     uint32_t rows, uint32_t cols,
		     CuStorageOrder order)
	: _handle(handle), _dMat(NULL), _rows(rows), _cols(cols), _storageOrder(order)
{
	_refCt = new uint32_t(1);
	
	AllocateMatrix();
}

CuMat::CuMat(const CuMat &other)
	: _handle(other._handle), _dMat(other._dMat), _rows(other._rows), _cols(other._cols),
	  _refCt(other._refCt), _storageOrder(other._storageOrder)
{
	// Increment the ref count
	++(*_refCt);
}

CuMat::~CuMat()
{
	// Decrement the ref count
	--(*_refCt);
	if (*_refCt == 0)
	{
		delete _refCt;
		FreeMatrix();
	}
}

bool CuMat::Empty() const
{
	return !_dMat || !_rows || !_cols;
}

bool CuMat::SingleOwner() const
{
	return *_refCt == 1;
}

CuMat &CuMat::operator=(CuMat other)
{
	swap(*this, other);
	return *this;
}

CuMat CuMat::Copy() const
{
	CuMat ret(_handle, _rows, _cols, _storageOrder);
	
	if (_dMat)
	{
		hipError_t status = hipMemcpy(ret._dMat, _dMat, _rows * _cols * sizeof(Real),
					   hipMemcpyDeviceToDevice);
		
		if (status != hipSuccess)
			throw runtime_error("Unable to copy the device memory from this matrix into the copy.");
	}
	
	return ret;
}

void CuMat::CopyToDevice(const Real *hMatrix)
{
	hipblasStatus_t status = hipblasSetMatrix(_rows, _cols, sizeof(Real),
											hMatrix, _rows, _dMat, _rows);
	
	if (status != HIPBLAS_STATUS_SUCCESS)
		throw runtime_error("Unable to copy the host matrix to the device.");
}

void CuMat::CopyToDevice(const CMatrix &hMatrix)
{
	assert(_rows == hMatrix.rows() &&
		   _cols == hMatrix.cols());
	
	CopyToDevice(hMatrix.data());
}

void CuMat::CopyToDevice(const RMatrix &hMatrix)
{
	CMatrix cMat = hMatrix;
	CopyToDevice(cMat);
}

void CuMat::CopyToDeviceAsync(const Real *hMatrix, hipStream_t stream)
{
	hipblasStatus_t status = hipblasSetMatrixAsync(_rows, _cols, sizeof(Real),
								hMatrix, _rows, _dMat, _rows, stream);
	
	if (status != HIPBLAS_STATUS_SUCCESS)
		throw runtime_error("Unable to copy the host matrix to the device.");
}

void CuMat::CopyToDeviceAsync(const CMatrix &hMatrix, hipStream_t stream)
{
	assert(_rows == hMatrix.rows() &&
		   _cols == hMatrix.cols());
	
	CopyToDeviceAsync(hMatrix.data(), stream);
}

void CuMat::CopyToDeviceAsync(const RMatrix &hMatrix, hipStream_t stream)
{
	CMatrix cMat = hMatrix;
	CopyToDeviceAsync(cMat, stream);
}

void CuMat::CopyToHost(Real* hMatrix) const
{
	hipblasStatus_t status = hipblasGetMatrix(_rows, _cols, sizeof(Real),
											_dMat, _rows, hMatrix, _rows);

	if (status != HIPBLAS_STATUS_SUCCESS)
		throw runtime_error("Unable to copy the device matrix to the host.");
}

void CuMat::CopyToHost(CMatrix& hMatrix) const
{
	CopyToHost(hMatrix.data());
}

void CuMat::CopyToHost(RMatrix& hMatrix) const
{
	CMatrix cMat(hMatrix.rows(), hMatrix.cols());
	CopyToHost(cMat);
	hMatrix = cMat;
}

void CuMat::CopyToHostAsync(Real* hMatrix, hipStream_t stream)
{
	hipblasStatus_t status = hipblasGetMatrixAsync(_rows, _cols, sizeof(Real),
								_dMat, _rows, hMatrix, _rows, stream);

	if (status != HIPBLAS_STATUS_SUCCESS)
		throw runtime_error("Unable to copy the device matrix to the host.");
}

void CuMat::CopyToHostAsync(CMatrix& hMatrix, hipStream_t stream)
{
	CopyToHostAsync(hMatrix.data(), stream);
}

CuMat operator+(const CuMat &a, const CuMat &b)
{
	CuMat ret;
	a.BinaryExpr<false>(b, ret, CuPlus());
	return ret;
}
CuMat operator-(const CuMat &a, const CuMat &b)
{
	CuMat ret;
	a.BinaryExpr<false>(b, ret, CuMinus());
	return ret;
}
CuMat operator*(const CuMat &a, const CuMat &b)
{
	static const float s_default = 1.0f;

	// Make sure the matrices are valid
	assert(a._cols == b._rows);
	assert(!a.Empty() && !b.Empty());
	assert(a._handle == b._handle);

	CuMat ret(a._handle, a._rows, b._cols);

	hipblasStatus_t status =
			hipblasSgemm(a._handle, a.GetTransOrder(), b.GetTransOrder(),
							a._rows, b._cols, a._cols,
							&s_default, a._dMat, a._rows,
							b._dMat, b._rows,
							NULL,
							NULL,
							0);

	if (status != HIPBLAS_STATUS_SUCCESS)
		throw runtime_error("The matrix multiplication failed.");

	return ret;
}

CuMat &operator+=(CuMat &a, const CuMat &b)
{
	a.BinaryExpr(b, CuPlus());
	return a;
}
CuMat &operator-=(CuMat &a, const CuMat &b)
{
	a.BinaryExpr(b, CuMinus());
	return a;
}

void CuMat::CoeffMultiply(Real val)
{
	CoeffMultiply(val, *this);
}

void CuMat::CoeffMultiply(Real val, CuMat& dest) const
{
	UnaryExpr<false>(dest, CuUnaryScale(val));
}

void CuMat::CoeffMultiply(const CuMat& b)
{
	CoeffMultiply(b, *this);
}

void CuMat::CoeffMultiply(const CuMat& b, CuMat& dest) const
{
	BinaryExpr<false>(b, dest, CuMultiply());
}

void CuMat::AddScaled(Real scaleThis, const CuMat& b, Real scaleB)
{
	AddScaled(scaleThis, b, scaleB, *this);
}

void CuMat::AddScaled(Real scaleThis, const CuMat& b, Real scaleB,
		CuMat& dest) const
{
	AssertSameDims(b);

	BinaryExpr<false>(b, dest, CuAddScaledBinary(scaleThis, scaleB));
}

void CuMat::Resize(uint32_t rows, uint32_t cols)
{
	// Test for a no-op
	if (SingleOwner() && _rows == rows && _cols == cols)
		return;

	// Ensure exclusive ownership of the matrix before
	// modifying it
	PrepareForWrite(false);

	_rows = rows;
	_cols = cols;

	// Free the old buffer if it is valid
	FreeMatrix();

	// Allocate the new matrix of the specified size
	AllocateMatrix();
}

void CuMat::ResizeLike(const CuMat& like)
{
	Resize(like._rows, like._cols);
}

void CuMat::Reshape(uint32_t rows, uint32_t cols)
{
	throw runtime_error("Not implemented.");
}

void CuMat::PrepareForWrite(bool alloc)
{
	// This is a copy on modify paradigm,
	// so if this instance is a sole owner of the data,
	// then nothing needs to be done
	if (*_refCt == 1)
		return;

	_refCt = new uint32_t(1);

	if (alloc)
		AllocateMatrix();
	else
		_dMat = NULL;
}

void CuMat::AllocateMatrix()
{
	_dMat = NULL;

	if (_rows == 0 || _cols == 0)
		return;

	hipError_t cudaStat = hipMalloc(&_dMat, _rows * _cols * sizeof(Real));
	if (cudaStat != hipSuccess)
		throw runtime_error("Unable to allocate the specified matrix");
}

void CuMat::FreeMatrix()
{
	// Free the device memory
	hipFree(_dMat);
}



void CuMat::AssertSameDims(const CuMat& other) const
{
	if (_rows != other._rows)
		throw runtime_error("The specified matrix doesn't have the same number of rows as this one.");
	if (_cols != other._cols)
		throw runtime_error("The specified matrix doesn't have the same number of columns as this one.");
}

CuStorageOrder CuMat::InverseOrder(CuStorageOrder order)
{
	switch (order)
	{
	case CuColMajor:
		return CuRowMajor;
	case CuRowMajor:
		return CuColMajor;
	default:
		throw runtime_error("Invalid storage order");
	}
}

hipblasOperation_t CuMat::GetTransOrder() const
{
	switch (_storageOrder)
	{
	case CuColMajor:
		return HIPBLAS_OP_N;
	case CuRowMajor:
		return HIPBLAS_OP_T;
	default:
		throw runtime_error("Invalid storage order");
	}
}

void swap(CuMat &a, CuMat &b)
{
	swap(a._handle, b._handle);
	swap(a._dMat, b._dMat);
	swap(a._rows, b._rows);
	swap(a._cols, b._cols);
	swap(a._refCt, b._refCt);
	swap(a._storageOrder, b._storageOrder);
}

CuScopedWeakTranspose::CuScopedWeakTranspose(CuMat& mat)
	: _mat(mat)
{
	Invert();
}

CuScopedWeakTranspose::~CuScopedWeakTranspose()
{
	// Undo the inversion
	Invert();
}

void CuScopedWeakTranspose::Invert()
{
	swap(_mat._rows, _mat._cols);

	_mat._storageOrder = CuMat::InverseOrder(_mat._storageOrder);
}
