/*
 * cublas_ut_helper.cu
 *
 *  Created on: May 30, 2014
 *      Author: mike
 */

#include "inc/cublas_ut_helper.cuh"

#include <stdexcept>

using namespace std;

hipblasHandle_t UTGetCublasHandle()
{
	static hipblasHandle_t s_handle = 0;

	hipError_t result = hipInit(0);

	if (!s_handle)
	{
		hipblasStatus_t /*status = cublasInit();*/

		status = hipblasCreate(&s_handle);

		if (status != HIPBLAS_STATUS_SUCCESS)
			throw runtime_error("Unable to allocate the cublas handle");
	}

	return s_handle;
}
